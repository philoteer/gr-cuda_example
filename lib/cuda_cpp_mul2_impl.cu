#include "hip/hip_runtime.h"
/* -*- c++ -*- */
/*
 * Copyright 2020 gr-cuda_example author.
 *
 * This is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation; either version 3, or (at your option)
 * any later version.
 *
 * This software is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.	See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this software; see the file COPYING.	If not, write to
 * the Free Software Foundation, Inc., 51 Franklin Street,
 * Boston, MA 02110-1301, USA.
 */

#ifdef HAVE_CONFIG_H
#include "config.h"
#endif

#include <gnuradio/io_signature.h>
#include "cuda_cpp_mul2_impl.cuh"

__global__ void cuda_kernel(float * in, float* out, size_t num_samples) {

	
	//get position
	const int pos = (threadIdx.x + (blockIdx.x * blockDim.x));
	
	if (pos < num_samples) 
	{
		out[pos*2] = in[pos*2] * 2;	//I
		out[pos*2+1] = in[pos*2+1] * 2;	//Q (you can use hip/hip_complex.h also; with an assumption that gr_complex and hipFloatComplex uses similar data structure (I/Q interleaved)
	}


}

namespace gr {
	namespace cuda_example {

		cuda_cpp_mul2::sptr
		cuda_cpp_mul2::make(int device_num, int vlen)
		{
			return gnuradio::get_initial_sptr
				(new cuda_cpp_mul2_impl(device_num, vlen));
		}


		/*
		 * The private constructor
		 */
		cuda_cpp_mul2_impl::cuda_cpp_mul2_impl(int device_num, int _vlen)
			: gr::sync_block("cuda_cpp_mul2",
							gr::io_signature::make(1, 1, sizeof(gr_complex) * _vlen),
							gr::io_signature::make(1, 1, sizeof(gr_complex) * _vlen))
		{
			vlen = _vlen;
			threads.x = 512;	//change ths as needed
			threads.y = 1;
			
			blocks.x = (vlen) / (threads.x);
			threads.y = 1;

			//set CUDA device
			cudaStatus = hipSetDevice(device_num);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipSetDevice failed!	Do you have a CUDA-capable GPU installed?");

			}

			// Allocate GPU buffers for the input	and output arrays
			cudaStatus = hipMalloc((void**)&dev_input,	vlen * sizeof(gr_complex));
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMalloc failed!");

			}

			cudaStatus = hipMalloc((void**)&dev_output, vlen * sizeof(gr_complex));
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMalloc failed!");

			}


		}

		/*
		 * Our virtual destructor.
		 */
		cuda_cpp_mul2_impl::~cuda_cpp_mul2_impl()
		{
		}

		int
		cuda_cpp_mul2_impl::work(int noutput_items,
				gr_vector_const_void_star &input_items,
				gr_vector_void_star &output_items)
		{
			//*in: input data buffer; *out: output data buffer; ring-buffer like implementation (internally)
			const gr_complex *in = (const gr_complex *) input_items[0];
			gr_complex *out = (gr_complex *) output_items[0];

			//for each vector:
			for(int i=0; i < noutput_items; i++)
			{
				//copy data to CUDA device
				cudaStatus = hipMemcpy(dev_input, (in+i*vlen), vlen * sizeof(gr_complex), hipMemcpyHostToDevice);
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "hipMemcpy failed!");
				}
				
				//run the CUDA kernel
				cuda_kernel << <blocks, threads >> > (dev_input, dev_output, vlen);
					
				//handle errors
				cudaStatus = hipGetLastError();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
				}
				
				// hipDeviceSynchronize waits for the kernel to finish, and returns
				// any errors encountered during the launch.
				cudaStatus = hipDeviceSynchronize();
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
				}
				
				//copy data from the CUDA device back to GNU Radio
				cudaStatus = hipMemcpy((out+i*vlen), dev_output, vlen * sizeof(gr_complex), hipMemcpyDeviceToHost);
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "hipMemcpy failed!");
				}

			}
			
			//clean-up
			return noutput_items;

		}

	} /* namespace cuda_example */
} /* namespace gr */

